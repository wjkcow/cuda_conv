#include "hip/hip_runtime.h"
#include "conv_lib.h"

// start_s is included but end_s is not
// launch this kernel with grid of kernel_stack * img_stack  (96 *5000)

// img[stack_num][channel][img_y][img_x]                                          5000 3 96 96
// kernel[kernel_stack_num][channel][kernel_y][kernex_x]                          96   3 7  7
// result[stack_num][kernel_stack_num][img_y-kernel_y + 1][img_x - kernel_x + 1]  5000 96 90 90
__global__ void valid_conv_kernel(float * img_base, float * filter_base, float * result_base,
	int img_x_dim, int img_y_dim, int channel,
	int filter_x_dim, int filter_y_dim){

	int img_stack      =  blockIdx.x;  // 5000

	int filter_stack   =  blockIdx.y; // 96
	int filter_stack_dim = blockDim.y;


	int result_x_dim = img_x_dim - filter_x_dim + 1;
	int result_y_dim = img_y_dim - filter_y_dim + 1;

	float * my_img_base = img_base + img_stack * img_x_dim * img_y_dim * channel;
	float * my_filter_base = filter_base + channel * filter_x_dim * filter_y_dim;
	float * my_result_base = result_base + img_stack * filter_stack_dim * result_x_dim * result_y_dim 
	                         + filter_stack * result_x_dim + result_y_dim;

#define result_v2(Y, X)     *(my_result_base + ( Y ) * result_x_dim + X)
#define filter_v3(C, Y, X)  *(my_filter_base + ( C )* filter_x_dim * filter_y_dim + ( Y ) * filter_x_dim + (X) )
#define img_v3(C, Y, X) 	*(my_img_base + ( C ) * img_y_dim * img_x_dim + ( Y )* img_x_dim + (X) )

	// int line_num = threadIdx.x;
	// int result_y_start =  line_num * line_step;
	// int result_y_end   =  line_num * (line_step + 1)
	for(int result_y = 0; result_y < result_y_dim; ++ result_y){
		for(int result_x = 0; result_x < result_x_dim; ++ result_x){
			float result = 0;
			for (int filter_c = 0; filter_c < channel; ++filter_c)
			{
				for (int filter_y = 0; filter_y < channel; ++filter_y){
					for (int filter_x = 0; filter_x < channel; ++filter_x){
						result += filter_v3(filter_c, filter_y, filter_x) * img_v3(filter_c, result_y_dim + filter_y, result_x_dim + filter_x);
					}
				}
			}
			result_v2(result_y, result_x) = result;
		}
	}


}



Status cuda_conv(Image img, Kernel ker, float *result){
	float  *img_base, *ker_base, *result_base;
	int img_size = img.image_x_dim * img.image_y_dim * img.image_channel * img.image_stack_num;
	int ker_size = ker.kernel_x_dim * ker.kernel_y_dim * ker.kernel_channel * ker.kernel_stack_num;
	int result_size = ker.kernel_stack_num * img.image_stack_num * (img.image_x_dim  - ker.kernel_x_dim + 1) * (img.image_y_dim - ker.kernel_y_dim + 1);

	hipMalloc((void **) & img_base, img_size*sizeof(float));
	hipMalloc((void **) & ker_base, ker_size*sizeof(float));
	hipMalloc((void **) & result_base, result_size*sizeof(float));
	hipMemcpy(img_base, img.image_base, img_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(ker_base, ker.kernel_base, ker_size*sizeof(float), hipMemcpyHostToDevice);

	int gridX = img.image_stack_num;
	int gridY = ker.kernel_stack_num;
	int threadX = 1;

	dim3 grid(gridX, gridY);
	dim3 block(threadX);
	valid_conv_kernel<<<grid, block, 0, 0>>>(img_base, ker_base, result_base, img.image_x_dim, img.image_y_dim, img.image_channel, 
		ker.kernel_x_dim, ker.kernel_y_dim);

	hipMemcpy(result, result_base, result_size*sizeof(float), hipMemcpyDeviceToHost);
	Status s;
	s.msg_len = 0;
	s.error = ALL_RIGHT;
	return s;

}

